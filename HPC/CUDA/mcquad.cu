#include "hip/hip_runtime.h"
/*                                                                |
 *              MCQUAD - CUDA MONTE CARLO INTEGRATOR              |
 *________________________________________________________________|
 *
 *      Performs a Monte Carlo integration of the form 
 *      \int_{0}^{\inf} \exp{-x} g(x) dx for g(x) = ln(x) 
 *      
 *      Draws N samples for each integration according to the density
 *      -> g(x).
 *
 *      INPUT PARAMETERS: 
 *	ntrials (total nuber of trials),
 *      nsamps  (samples taken per trial)
 *
 * Written by Brandon B. Miller
 */



#include <stdlib.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>


__global__ void setup_kernel(int N, long int seed, hiprandState_t *state) {
	// Set up the RNG for each sample thread	
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < N) {
		// Each RNG state is different
		// It augomatically increments itself
		hiprand_init(seed, id, 0, &state[id]);
	}
}


__global__ void do_trials(int ntrials, int nsamps, double* results, hiprandState_t *state) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	// We will have each thread do a trial. So we need to launch 
	// Ntrials blocks each with one thread.  
	if (id < ntrials) {
		double sum = 0;	
		for (int sample = 0; sample < nsamps; sample++) { 
			sum += cos(-log(hiprand_uniform_double(&state[id]))); 	

		}
		results[id] = sum / nsamps; // Answer
	}
}


static void HandleError (hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(1);
	}
}
#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))


int main(int argc, char* argv[]) {

	// Initial Machinery to select the GPU
	// ___________________________________ 
	hipDeviceProp_t prop; // This is a blank struct at this point	
	int dev;	     
	memset(&prop, 0, sizeof(hipDeviceProp_t)); // Initialize the struct

	prop.multiProcessorCount = 13;
	hipChooseDevice(&dev, &prop);
	hipSetDevice(dev);
	hipGetDeviceProperties(&prop, dev);


	// ___________________________________


	// Initial Machinery to read in params
	// __________________________________

	float tym;	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop );
	hipEventRecord(start, 0);
	int nsamps;
	int ntrials;
	FILE* inputfile;
	FILE* outputfile;
	
	if (argc != 3) {
		printf("Incorrect usage: only enter the input and output filenames\n");
		return 0;
	}
	inputfile = fopen(argv[1], "r");
	if (!inputfile) {
		printf("Unable to open input file \n");
		return 0;
	}
	fscanf(inputfile, "%d", &nsamps);
	fscanf(inputfile, "%d", &ntrials);
	// __________________________________

	double* results = (double *)malloc(ntrials * sizeof(double));	

	// Random number generation
	hiprandState_t* dev_states;	
	double* dev_results; // will contain final random numbers
	
	if ( hipSuccess != hipMalloc((void**)&dev_results, ntrials*sizeof(double)) ) {
		printf("hipMalloc Failed...");
		exit(1);
	}
	// THERE IS NOW AN NTRIALS LENGTH ARRAY IN GLOBAL MEM ON THE DEVICE 
	if ( hipSuccess != hipMalloc((void**)&dev_states, ntrials*sizeof(hiprandState_t)) ) {
		printf("hipMalloc Failed...");
		exit(1);
	}
	// dev_states is an array containing an RNG state to be used for each trial
	// We will index into it uniquely based on thread and blockID within the kernel
	setup_kernel<<<ntrials, 1>>>(nsamps, time(NULL), dev_states);
	
	// FIXME - Launch a block for each trial with one thread each - SLOW	
	do_trials<<<ntrials, 1>>>(ntrials, nsamps, dev_results, dev_states); 

	// Retrieve results
	hipMemcpy(results, dev_results, ntrials*sizeof(double), hipMemcpyDeviceToHost);

	outputfile = fopen(argv[2], "wb");
	fwrite(results, sizeof(double), ntrials, outputfile);

	// Closing machinery
	hipFree(dev_results);
	hipFree(dev_states);
	free(results);
	fclose(outputfile);
	fclose(inputfile);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tym, start, stop);
	printf("Elapsed time %3.1f milliseconds", tym);
	return 0;
}
