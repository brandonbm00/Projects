#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

static void HandleError (hipError_t err, const char* file, int line) {
        if (err != hipSuccess) {
                printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
                exit(1);
        }
}
#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))





void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
    return;
}


// Kernel function

__global__ void mmult(double a[], double b[], double c[], a_n, a_m, b_n, b_m) {
    extern __shared__ double local[];

    int CX   = blockIdx.x;
    int CY   = blockIdx.y;
    int X    = threadIdx.x;
    int REM  = a_m % blockDim.x; 
    int ITER = (a_m - REM) / blockDim.x; 
    
    double my_row = &a[] 

    for (int i = 0; i < ITER; i++) {
        local[X] = 
    }
}

int main(int argc, char* argv[]) {
    // Initial Machinery to select the GPU
    // ___________________________________ 
    hipDeviceProp_t prop; // This is a blank struct at this point    
    int dev;
    memset(&prop, 0, sizeof(hipDeviceProp_t)); // Initialize the struct

    prop.multiProcessorCount = 13;
    hipChooseDevice(&dev, &prop);
    HANDLE_ERROR(hipSetDevice(dev));
    hipGetDeviceProperties(&prop, dev);
   
     
    printf("Device Properties: -------------------------------- \n");
    printDevProp(prop);
    printf("End Device Properties: ---------------------------- \n");
 
    // Get dimensions of matrices from command line


    printf("Begin. --------------------------------------------\n");  
    if (argc < 4) {
       printf("Please enter four args: A_n, A_m, B_n, B_m \n");
       return 1;
    }

    int A_n = atoi(argv[1]);
    int A_m = atoi(argv[2]);
    int B_n = atoi(argv[3]);
    int B_m = atoi(argv[4]);

    if (A_m != B_n) {
        printf("Inner matrix dimensions A_m = %d and B_n = %d must match \n", A_m, B_n);
        return 2;
    }

    printf("Matrices will be A = (%d x %d), B = (%d x %d) \n", A_n, A_m, B_n, B_m);
    printf("Final matrix will be C = (%d x %d) \n", A_n, B_m);

    // instantiate matrices
    
    double* A = (double *)malloc(A_n*A_m*sizeof(double));
    double* B = (double *)malloc(B_n*B_m*sizeof(double));

    double* A_gpu;
    double* B_gpu;


    for (int i = 0; i < A_n*A_m; i++) {
        A[i] = double(i);  
    }
    for (int i = 0; i < B_n*B_m; i++) {
        B[i] = double(i);  
    }
     


    // Copy data down to GPU
    if ( hipSuccess != hipMalloc((void**)&A_gpu, A_n*A_m*sizeof(double)) ) {
        printf("hipMalloc Failed...\n");
        exit(1);
    }
    if ( hipSuccess != hipMalloc((void**)&B_gpu, B_n*B_m*sizeof(double)) ) {
        printf("hipMalloc Failed...\n");
        exit(1);
    }

    hipMemcpy(A_gpu, A, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, B_n*B_m*sizeof(double), hipMemcpyHostToDevice);



    free(A);
    free(B);
    hipFree(A_gpu);
    hipFree(B_gpu);
    return 0;
}
